#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <sstream>

#include <boost/array.hpp>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "born_wolf_psf_table.hpp"

#define PIXELS 63
#define BLOCKS 63 * 63
#define THREADS 128
#define N_DIV 11

typedef struct
{
    double px;
    double py;
    double xmin;
    double ymin;
    unsigned int N;
    unsigned int M;
    double imin;
    double jmin;
    unsigned int N_pixel;
    double pixel_length;
    double delta_length;
    double alpha_factor;
    double psi_factor;
    double C;
} parameter_type;

template <unsigned int BLOCK_SIZE>
__global__ void born_wolf_psf_tbl_gpu_kernel(
    double* d_y, double* d_data, const parameter_type params)
{
    extern __shared__ double s_x[];
    const unsigned int tid = threadIdx.x;
    // const unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
	// const unsigned int idx = bid * blockDim.x + tid;

    if (tid < N_DIV * N_DIV)
    {
        const double xmin = params.xmin + params.pixel_length * blockIdx.x;
        const double ymin = params.ymin + params.pixel_length * blockIdx.y;
        // const double xmax = xmin + params.pixel_length;
        // const double ymax = ymin + params.pixel_length;

        const unsigned int xidx = tid % N_DIV;
        const unsigned int yidx = tid / N_DIV;

        const double x = xmin + params.delta_length * xidx;
        const double y = ymin + params.delta_length * yidx;
        const double dx = x - params.px;
        const double dy = y - params.py;
        const double r = sqrt(dx * dx + dy * dy);

        const double n1 = r * params.alpha_factor;
        const double n2 = floor(n1);
        const double m1 = 0.0 * params.psi_factor;
        const double m2 = floor(m1);
        const unsigned int n = static_cast<unsigned int>(n2);
        const unsigned int m = static_cast<unsigned int>(m2);

        if (n < params.N && m < params.M)
        {
            const double pr = n1 - n2;
            const double pz = m1 - m2;
            const double v00 = d_y[m * params.N + n];
            const double v10 = d_y[m * params.N + n + 1];
            const double v01 = d_y[(m + 1) * params.N + n];
            const double v11 = d_y[(m + 1) * params.N + n + 1];
            const double I = (1.0 - pr) * ((1.0 - pz) * v00 + pz * v01) + pr * ((1.0 - pz) * v10 + pz * v11);

            const double factor1 = ((xidx == 0 || xidx == N_DIV - 1) ? 1.0 / 3.0 : (xidx % 2 == 0 ? 2.0 / 3.0 : 4.0 / 3.0));
            const double factor2 = ((yidx == 0 || yidx == N_DIV - 1) ? 1.0 / 3.0 : (yidx % 2 == 0 ? 2.0 / 3.0 : 4.0 / 3.0));

            s_x[tid] = factor1 * factor2 * params.delta_length * params.delta_length * I;
        }
    }

    __syncthreads();

    // for (unsigned int i = 1; i < blockDim.x; i *= 2)
    // {
    //     if (tid % (2 * i) == 0)
    //     {
    //         s_x[tid] += s_x[tid + i];
    //     }
    //     __syncthreads();
    // }

    // for (unsigned int i = blockDim.x / 2; i > 0; i >>= 1)
    // {
    //     if (tid < i)
    //     {
    //         s_x[tid] += s_x[tid + i];
    //     }
    //     __syncthreads();
    // }

    if (BLOCK_SIZE >= 512)
    {
        if (tid < 256)
        {
            s_x[tid] += s_x[tid + 256];
        }
        __syncthreads();
    }

    if (BLOCK_SIZE >= 256)
    {
        if (tid < 128)
        {
            s_x[tid] += s_x[tid + 128];
        }
        __syncthreads();
    }

    if (BLOCK_SIZE >= 128)
    {
        if (tid < 64)
        {
            s_x[tid] += s_x[tid + 64];
        }
        __syncthreads();
    }

    if (tid < 32)
    {
        if (BLOCK_SIZE >= 64) s_x[tid] += s_x[tid + 32];
        if (BLOCK_SIZE >= 32) if (tid < 16) s_x[tid] += s_x[tid + 16];
        if (BLOCK_SIZE >= 16) if (tid < 8) s_x[tid] += s_x[tid + 8];
        if (BLOCK_SIZE >= 8) if (tid < 4) s_x[tid] += s_x[tid + 4];
        if (BLOCK_SIZE >= 4) if (tid < 2) s_x[tid] += s_x[tid + 2];
        if (BLOCK_SIZE >= 2) if (tid < 1) s_x[tid] += s_x[tid + 1];
    }

    if (tid == 0)
    {
        const int i1 = static_cast<int>(static_cast<double>(blockIdx.x) + params.imin);
        const int j1 = static_cast<int>(static_cast<double>(blockIdx.y) + params.jmin);
        if (0 <= i1 && i1 < params.N_pixel && 0 <= j1 && j1 < params.N_pixel)
        {
            d_data[static_cast<unsigned int>(j1 * params.N_pixel + i1)] += s_x[0] * params.C;
        }
    }
}

/*
 * const unsigned int N_pixel = 600;
 * const double objective = 100.0;
 * const double pixel_length = 6500.0 / objective;
 * const double k = 2.0 * M_PI / 508.0;
 * const double N_A = 1.4;
 * const unsigned int N_point = 1000;
 */

void born_wolf_psf_tbl_gpu(
    double data[], unsigned int const N_pixel, double const pixel_length,
    double points[][3], unsigned int const N_point,
    double const I, double c[3], double const k, double const N_A,
    double const cutoff)
{
    const unsigned int N = born_wolf_psf_table::born_wolf_psf_table.N;
    const unsigned int M = born_wolf_psf_table::born_wolf_psf_table.M;
    const double rmax = born_wolf_psf_table::born_wolf_psf_table.rmax;
    const double zmax = born_wolf_psf_table::born_wolf_psf_table.zmax;

    const double alpha = N_A * k;
    const double C = alpha * alpha / M_PI;
    const double alpha_factor = alpha * N / rmax;
    const double psi_factor = 0.5 * alpha * N_A * M / zmax;
    const double offset = N_pixel * pixel_length * -0.5;
    const double delta_length = pixel_length / (N_DIV - 1);

    const unsigned int tbl_size = N * M; // 802401;
    thrust::host_vector<double> y(tbl_size);
    memcpy((void *)thrust::raw_pointer_cast(y.data()),
           (const void *)born_wolf_psf_table::born_wolf_psf_table.y,
           tbl_size * sizeof(double));
    thrust::device_vector<double> d_y = y;

    thrust::device_vector<double> d_x(N_pixel * N_pixel, 0.0);
    // thrust::host_vector<double> x(N_pixel * N_pixel, 0.0);

    for (unsigned int cnt = 0; cnt < N_point; ++cnt)
    {
        const double x = points[cnt][0] - c[0];
        const double y = points[cnt][1] - c[1];

        const double i0 = floor((x - offset) / pixel_length);
        const double j0 = floor((y - offset) / pixel_length);
        const double imin = i0 - floor(PIXELS * 0.5);
        const double jmin = j0 - floor(PIXELS * 0.5);
        const double xmin = imin * pixel_length + offset;
        const double ymin = jmin * pixel_length + offset;

        parameter_type params = {
            x, y, xmin, ymin,
            N, M, imin, jmin, N_pixel,
            pixel_length, delta_length,
            alpha_factor, psi_factor, C};

        born_wolf_psf_tbl_gpu_kernel<THREADS><<<dim3(PIXELS, PIXELS, 1), THREADS, THREADS * sizeof(double)>>>(thrust::raw_pointer_cast(d_y.data()), thrust::raw_pointer_cast(d_x.data()), params);
    }

    // x = d_x;
    hipMemcpy(data, thrust::raw_pointer_cast(d_x.data()), (N_pixel * N_pixel) * sizeof(double), hipMemcpyDeviceToHost);
}

std::string point_as_str(double p[3])
{
    std::stringstream sout;
    sout << std::showpos;
    sout << "(" << p[0] << "," << p[1] << "," << p[2] << ")";
    return sout.str();
}

void read_input(
    char const filename[], double points[][3], unsigned int const data_size,
    double shift[3], double const scale)
{
    std::ifstream fin(filename);
    std::string buf;

    std::getline(fin, buf); // Ignore a header line

    unsigned int i(0);
    while (fin && std::getline(fin, buf) && i < data_size)
    {
        std::string token;
        std::istringstream stream(buf);
        std::stringstream ss;
        double val;
        unsigned int sid;

        double x, y, z;

        {
            std::getline(stream, token, ',');
            ss << token;
            ss >> val;
            // t = val;
            ss.str("");
            ss.clear(std::stringstream::goodbit);
        }
        {
            std::getline(stream, token, ',');
            ss << token;
            ss >> val;
            x = val;
            ss.str("");
            ss.clear(std::stringstream::goodbit);
        }
        {
            std::getline(stream, token, ',');
            ss << token;
            ss >> val;
            y = val;
            ss.str("");
            ss.clear(std::stringstream::goodbit);
        }
        {
            std::getline(stream, token, ',');
            ss << token;
            ss >> val;
            z = val;
            ss.str("");
            ss.clear(std::stringstream::goodbit);
        }
        {
            std::getline(stream, token, ',');
            std::getline(stream, token, ',');
            ss << token;
            ss >> sid;
            ss.str("");
            ss.clear(std::stringstream::goodbit);
        }

        // if (sid != 0)
        // {
        //     continue;
        // }

        points[i][0] = (y - shift[0]) * scale;
        points[i][1] = (z - shift[1]) * scale;
        points[i][2] = fabs(x - shift[2]) * scale;
        // intensity[i] = emission(points[i][2]);
        // std::cout << point_as_str(points[i]) << std::endl;
        i++;
    };
}

int main(int argc, char** argv)
{
    const unsigned int N_point(1000);
    double points[N_point][3];
    double shift[3] = {15e-6, 15e-6, 5.06228e-07};
    double scale = 1000.0 * 1e+6;

    const double lambda = 508.0; // nm
    const double N_A = 1.4;
    const double k = 2 * M_PI / lambda;
    double focal_point[] = {0, 0, 0};
    const double cutoff = 2000.0;
    const unsigned int N_pixel = 600;
    const double objective = 100.0;
    const double pixel_length = 6500 / objective;

    boost::array<double, N_pixel * N_pixel> data;
    data.fill(0.0);

    for (unsigned int idx = 1; idx < argc; ++idx)
    {
        const std::string filename(argv[idx]);
        read_input(filename.c_str(), points, N_point, shift, scale);

        const double intensity = 1.0;

        born_wolf_psf_tbl_gpu(
            data.data(), N_pixel, pixel_length, points, N_point,
            intensity, focal_point, k, N_A, cutoff);
    }

    std::cout << std::scientific << std::setprecision(16);
    for (unsigned int i = 0; i < N_pixel * N_pixel; ++i)
    {
        std::cout << data[i] << std::endl;
    }

    return 0;
}
