#include "hip/hip_runtime.h"
#include "microscope_gpu.h"

extern "C" void born_wolf_psf_tbl_gpu(
    double data[], unsigned int const N_pixel, double const pixel_length,
    double points[][3], unsigned int const N_point,
    double const I, double c[3], double const k, double const N_A,
    double const cutoff);

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "born_wolf_psf_table.hpp"

#define PIXELS 63
#define BLOCKS 63 * 63
#define THREADS 128
#define N_DIV 11

typedef struct
{
    double px;
    double py;
    double xmin;
    double ymin;
    unsigned int N;
    unsigned int M;
    double imin;
    double jmin;
    unsigned int N_pixel;
    double pixel_length;
    double delta_length;
    double alpha_factor;
    double psi_factor;
    double C;
} parameter_type;

template <unsigned int BLOCK_SIZE>
__global__ void born_wolf_psf_tbl_gpu_kernel(
    double* d_y, double* d_data, const parameter_type params)
{
    extern __shared__ double s_x[];
    const unsigned int tid = threadIdx.x;
    // const unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
	// const unsigned int idx = bid * blockDim.x + tid;

    if (tid < N_DIV * N_DIV)
    {
        const double xmin = params.xmin + params.pixel_length * blockIdx.x;
        const double ymin = params.ymin + params.pixel_length * blockIdx.y;
        // const double xmax = xmin + params.pixel_length;
        // const double ymax = ymin + params.pixel_length;

        const unsigned int xidx = tid % N_DIV;
        const unsigned int yidx = tid / N_DIV;

        const double x = xmin + params.delta_length * xidx;
        const double y = ymin + params.delta_length * yidx;
        const double dx = x - params.px;
        const double dy = y - params.py;
        const double r = sqrt(dx * dx + dy * dy);

        const double n1 = r * params.alpha_factor;
        const double n2 = floor(n1);
        const double m1 = 0.0 * params.psi_factor;
        const double m2 = floor(m1);
        const unsigned int n = static_cast<unsigned int>(n2);
        const unsigned int m = static_cast<unsigned int>(m2);

        if (n < params.N && m < params.M)
        {
            const double pr = n1 - n2;
            const double pz = m1 - m2;
            const double v00 = d_y[m * params.N + n];
            const double v10 = d_y[m * params.N + n + 1];
            const double v01 = d_y[(m + 1) * params.N + n];
            const double v11 = d_y[(m + 1) * params.N + n + 1];
            const double I = (1.0 - pr) * ((1.0 - pz) * v00 + pz * v01) + pr * ((1.0 - pz) * v10 + pz * v11);

            const double factor1 = ((xidx == 0 || xidx == N_DIV - 1) ? 1.0 / 3.0 : (xidx % 2 == 0 ? 2.0 / 3.0 : 4.0 / 3.0));
            const double factor2 = ((yidx == 0 || yidx == N_DIV - 1) ? 1.0 / 3.0 : (yidx % 2 == 0 ? 2.0 / 3.0 : 4.0 / 3.0));

            s_x[tid] = factor1 * factor2 * params.delta_length * params.delta_length * I;
        }
    }

    __syncthreads();

    // for (unsigned int i = 1; i < blockDim.x; i *= 2)
    // {
    //     if (tid % (2 * i) == 0)
    //     {
    //         s_x[tid] += s_x[tid + i];
    //     }
    //     __syncthreads();
    // }

    // for (unsigned int i = blockDim.x / 2; i > 0; i >>= 1)
    // {
    //     if (tid < i)
    //     {
    //         s_x[tid] += s_x[tid + i];
    //     }
    //     __syncthreads();
    // }

    if (BLOCK_SIZE >= 512)
    {
        if (tid < 256)
        {
            s_x[tid] += s_x[tid + 256];
        }
        __syncthreads();
    }

    if (BLOCK_SIZE >= 256)
    {
        if (tid < 128)
        {
            s_x[tid] += s_x[tid + 128];
        }
        __syncthreads();
    }

    if (BLOCK_SIZE >= 128)
    {
        if (tid < 64)
        {
            s_x[tid] += s_x[tid + 64];
        }
        __syncthreads();
    }

    if (tid < 32)
    {
        if (BLOCK_SIZE >= 64) s_x[tid] += s_x[tid + 32];
        if (BLOCK_SIZE >= 32) if (tid < 16) s_x[tid] += s_x[tid + 16];
        if (BLOCK_SIZE >= 16) if (tid < 8) s_x[tid] += s_x[tid + 8];
        if (BLOCK_SIZE >= 8) if (tid < 4) s_x[tid] += s_x[tid + 4];
        if (BLOCK_SIZE >= 4) if (tid < 2) s_x[tid] += s_x[tid + 2];
        if (BLOCK_SIZE >= 2) if (tid < 1) s_x[tid] += s_x[tid + 1];
    }

    if (tid == 0)
    {
        const int i1 = static_cast<int>(static_cast<double>(blockIdx.x) + params.imin);
        const int j1 = static_cast<int>(static_cast<double>(blockIdx.y) + params.jmin);
        if (0 <= i1 && i1 < params.N_pixel && 0 <= j1 && j1 < params.N_pixel)
        {
            d_data[static_cast<unsigned int>(j1 * params.N_pixel + i1)] += s_x[0] * params.C;
        }
    }
}

/*
 * const unsigned int N_pixel = 600;
 * const double objective = 100.0;
 * const double pixel_length = 6500.0 / objective;
 * const double k = 2.0 * M_PI / 508.0;
 * const double N_A = 1.4;
 * const unsigned int N_point = 1000;
 */

void born_wolf_psf_tbl_gpu(
    double data[], unsigned int const N_pixel, double const pixel_length,
    double points[][3], unsigned int const N_point,
    double const I, double c[3], double const k, double const N_A,
    double const cutoff)
{
    const unsigned int N = born_wolf_psf_table::born_wolf_psf_table.N;
    const unsigned int M = born_wolf_psf_table::born_wolf_psf_table.M;
    const double rmax = born_wolf_psf_table::born_wolf_psf_table.rmax;
    const double zmax = born_wolf_psf_table::born_wolf_psf_table.zmax;

    const double alpha = N_A * k;
    const double C = alpha * alpha / M_PI;
    const double alpha_factor = alpha * N / rmax;
    const double psi_factor = 0.5 * alpha * N_A * M / zmax;
    const double offset = N_pixel * pixel_length * -0.5;
    const double delta_length = pixel_length / (N_DIV - 1);

    const unsigned int tbl_size = N * M; // 802401;
    thrust::host_vector<double> y(tbl_size);
    memcpy((void *)thrust::raw_pointer_cast(y.data()),
           (const void *)born_wolf_psf_table::born_wolf_psf_table.y,
           tbl_size * sizeof(double));
    thrust::device_vector<double> d_y = y;

    thrust::device_vector<double> d_x(N_pixel * N_pixel, 0.0);
    // thrust::host_vector<double> x(N_pixel * N_pixel, 0.0);

    for (unsigned int cnt = 0; cnt < N_point; ++cnt)
    {
        const double x = points[cnt][0] - c[0];
        const double y = points[cnt][1] - c[1];

        const double i0 = floor((x - offset) / pixel_length);
        const double j0 = floor((y - offset) / pixel_length);
        const double imin = i0 - floor(PIXELS * 0.5);
        const double jmin = j0 - floor(PIXELS * 0.5);
        const double xmin = imin * pixel_length + offset;
        const double ymin = jmin * pixel_length + offset;

        parameter_type params = {
            x, y, xmin, ymin,
            N, M, imin, jmin, N_pixel,
            pixel_length, delta_length,
            alpha_factor, psi_factor, C};

        born_wolf_psf_tbl_gpu_kernel<THREADS><<<dim3(PIXELS, PIXELS, 1), THREADS, THREADS * sizeof(double)>>>(thrust::raw_pointer_cast(d_y.data()), thrust::raw_pointer_cast(d_x.data()), params);
    }

    // x = d_x;
    hipMemcpy(data, thrust::raw_pointer_cast(d_x.data()), (N_pixel * N_pixel) * sizeof(double), hipMemcpyDeviceToHost);
}
